#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

__device__ float L[4002][4002];
__device__ float b[4002];
__device__ float x[4002];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(int i){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(i-1) ) )return;
	x[i] = x[i] - L[i][j] * x[j];
}

__global__ void kernel_2(){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=(4000-1) ) )return;
	x[i] = b[i];
	int thread_count_1 = (i-1)-0+1;
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>(i);
	hipDeviceSynchronize();
	x[i] = x[i] / L[i][i];
}

__global__ void main_kernel(){
	int thread_count_2 = (4000-1)-0+1;
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>();
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_L = (float*) malloc(sizeof(float)* (4002)* (4002));
	hipMemcpyFromSymbol(h_L,HIP_SYMBOL(L),sizeof(float)* (4002)* (4002));
	float* h_b = (float*) malloc(sizeof(float)* (4002));
	hipMemcpyFromSymbol(h_b,HIP_SYMBOL(b),sizeof(float)* (4002));
	float* h_x = (float*) malloc(sizeof(float)* (4002));
	hipMemcpyFromSymbol(h_x,HIP_SYMBOL(x),sizeof(float)* (4002));
	gettimeofday(&t2, 0);
	double time = 1.0*(t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time taken for execution is: %.8f sec\n", time);
	return 0;
}
