#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

__device__ float a[1602][2002];
__device__ float ab[1602][1802];
__device__ float abcd[1602][2202];
__device__ float b[2002][1802];
__device__ float c[1802][2402];
__device__ float cd[1802][2202];
__device__ float d[2402][2202];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(int i,int j,float* temp_1){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=(2000-1) ) )return;
	temp_1[k-0] = a[i][k] * b[k][j];
}

__global__ void kernel_2(int i){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(1800-1) ) )return;
	int thread_count_1 = (2000-1)-0+1;
	float* temp_1 = (float*)malloc(sizeof(float)*((2000-1)-0+1));
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>(i,j,temp_1);
	hipDeviceSynchronize();
	sumArray( temp_1,thread_count_1);
	hipDeviceSynchronize();
	ab[i][j] = temp_1[0];
}

__global__ void kernel_3(){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=(1600-1) ) )return;
	int thread_count_2 = (1800-1)-0+1;
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>(i);
	hipDeviceSynchronize();
}

__global__ void kernel_4(int i,int j,float* temp_2){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=(2400-1) ) )return;
	temp_2[k-0] = c[i][k] * d[k][j];
}

__global__ void kernel_5(int i){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(2200-1) ) )return;
	int thread_count_4 = (2400-1)-0+1;
	float* temp_2 = (float*)malloc(sizeof(float)*((2400-1)-0+1));
	kernel_4<<<ceil( 1.0 * thread_count_4/1024),1024>>>(i,j,temp_2);
	hipDeviceSynchronize();
	sumArray( temp_2,thread_count_4);
	hipDeviceSynchronize();
	cd[i][j] = temp_2[0];
}

__global__ void kernel_6(){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=(1800-1) ) )return;
	int thread_count_5 = (2200-1)-0+1;
	kernel_5<<<ceil( 1.0 * thread_count_5/1024),1024>>>(i);
	hipDeviceSynchronize();
}

__global__ void kernel_7(int i,int j,float* temp_3){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=(1800-1) ) )return;
	temp_3[k-0] = ab[i][k] * cd[k][j];
}

__global__ void kernel_8(int i){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(2200-1) ) )return;
	int thread_count_7 = (1800-1)-0+1;
	float* temp_3 = (float*)malloc(sizeof(float)*((1800-1)-0+1));
	kernel_7<<<ceil( 1.0 * thread_count_7/1024),1024>>>(i,j,temp_3);
	hipDeviceSynchronize();
	sumArray( temp_3,thread_count_7);
	hipDeviceSynchronize();
	abcd[i][j] = temp_3[0];
}

__global__ void kernel_9(){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=(1600-1) ) )return;
	int thread_count_8 = (2200-1)-0+1;
	kernel_8<<<ceil( 1.0 * thread_count_8/1024),1024>>>(i);
	hipDeviceSynchronize();
}

__global__ void main_kernel(){
	int thread_count_3 = (1600-1)-0+1;
	kernel_3<<<ceil( 1.0 * thread_count_3/1024),1024>>>();
	hipDeviceSynchronize();
	int thread_count_6 = (1800-1)-0+1;
	kernel_6<<<ceil( 1.0 * thread_count_6/1024),1024>>>();
	hipDeviceSynchronize();
	int thread_count_9 = (1600-1)-0+1;
	kernel_9<<<ceil( 1.0 * thread_count_9/1024),1024>>>();
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_a = (float*) malloc(sizeof(float)* (1602)* (2002));
	hipMemcpyFromSymbol(h_a,HIP_SYMBOL(a),sizeof(float)* (1602)* (2002));
	float* h_ab = (float*) malloc(sizeof(float)* (1602)* (1802));
	hipMemcpyFromSymbol(h_ab,HIP_SYMBOL(ab),sizeof(float)* (1602)* (1802));
	float* h_abcd = (float*) malloc(sizeof(float)* (1602)* (2202));
	hipMemcpyFromSymbol(h_abcd,HIP_SYMBOL(abcd),sizeof(float)* (1602)* (2202));
	float* h_b = (float*) malloc(sizeof(float)* (2002)* (1802));
	hipMemcpyFromSymbol(h_b,HIP_SYMBOL(b),sizeof(float)* (2002)* (1802));
	float* h_c = (float*) malloc(sizeof(float)* (1802)* (2402));
	hipMemcpyFromSymbol(h_c,HIP_SYMBOL(c),sizeof(float)* (1802)* (2402));
	float* h_cd = (float*) malloc(sizeof(float)* (1802)* (2202));
	hipMemcpyFromSymbol(h_cd,HIP_SYMBOL(cd),sizeof(float)* (1802)* (2202));
	float* h_d = (float*) malloc(sizeof(float)* (2402)* (2202));
	hipMemcpyFromSymbol(h_d,HIP_SYMBOL(d),sizeof(float)* (2402)* (2202));
	gettimeofday(&t2, 0);
	double time = 1.0*(t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time taken for execution is: %.8f sec\n", time);
	return 0;
}
