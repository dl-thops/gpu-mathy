#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

__device__ float A[123];
__device__ float B[123];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(){
	int i = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=i ) || !( i<=(120-1) ) )return;
	
	B[i] = 0.33333 * (A[i] + A[i-1] + A[i+1]);
}

__global__ void kernel_2(){
	int i = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=i ) || !( i<=(120-1) ) )return;
	
	A[i] = 0.33333 * (B[i] + B[i-1] + B[i+1]);
}

__global__ void kernel_3(){
	int t = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=t ) || !( t<=(40-1) ) )return;
	int thread_count_1 = (120-1)-1+1;
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>();
	hipDeviceSynchronize();
	int thread_count_2 = (120-1)-1+1;
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>();
	hipDeviceSynchronize();
}

__global__ void main_kernel(){
	int thread_count_3 = (40-1)-0+1;
	kernel_3<<<ceil( 1.0 * thread_count_3/1024),1024>>>();
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_A = (float*) malloc(sizeof(float)* (123));
	hipMemcpyFromSymbol(h_A,HIP_SYMBOL(A),sizeof(float)* (123));
	float* h_B = (float*) malloc(sizeof(float)* (123));
	hipMemcpyFromSymbol(h_B,HIP_SYMBOL(B),sizeof(float)* (123));
	gettimeofday(&t2, 0);
	double time = 1.0*(t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time taken for execution is: %.8f sec\n", time);
	return 0;
}
