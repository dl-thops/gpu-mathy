#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

__device__ float A[12003][12003];
__device__ float A_t[12002][12002];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(int i){
	int j = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=j ) || !( j<=(12000 - 2-1) ) )return;
	
	A_t[i][j] = (A[i][j] + A[i-1][j-1] + A[i-1][j] + A[i-1][j+1] + A[i][j-1] + A[i][j+1] + A[i+1][j-1] + A[i+1][j] + A[i+1][j+1])/9.0;
}

__global__ void kernel_2(){
	int i = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=i ) || !( i<=(12000 - 2-1) ) )return;
	int thread_count_1 = (12000 - 2-1)-1+1;
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>(i);
	hipDeviceSynchronize();
}

__global__ void kernel_3(int i){
	int j = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=j ) || !( j<=(12000 - 2-1) ) )return;
	A[i][j] = A_t[i][j];
}

__global__ void kernel_4(){
	int i = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=i ) || !( i<=(12000 - 2-1) ) )return;
	int thread_count_3 = (12000 - 2-1)-1+1;
	kernel_3<<<ceil( 1.0 * thread_count_3/1024),1024>>>(i);
	hipDeviceSynchronize();
}

__global__ void kernel_5(){
	int t = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=t ) || !( t<=(3000-1) ) )return;
	int thread_count_2 = (12000 - 2-1)-1+1;
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>();
	hipDeviceSynchronize();
	int thread_count_4 = (12000 - 2-1)-1+1;
	kernel_4<<<ceil( 1.0 * thread_count_4/1024),1024>>>();
	hipDeviceSynchronize();
}

__global__ void main_kernel(){
	int thread_count_5 = (3000-1)-0+1;
	kernel_5<<<ceil( 1.0 * thread_count_5/1024),1024>>>();
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_A = (float*) malloc(sizeof(float)* (12003)* (12003));
	hipMemcpyFromSymbol(h_A,HIP_SYMBOL(A),sizeof(float)* (12003)* (12003));
	float* h_A_t = (float*) malloc(sizeof(float)* (12002)* (12002));
	hipMemcpyFromSymbol(h_A_t,HIP_SYMBOL(A_t),sizeof(float)* (12002)* (12002));
	gettimeofday(&t2, 0);
	double time = 1.0*(t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time taken for execution is: %.8f sec\n", time);
	return 0;
}
