#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

__device__ float A[702][802][902];
__device__ float C4[902][902];
__device__ float sum_[902];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(int i,int j,int k,float* temp_1){
	int h = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=h ) || !( h<=(900-1) ) )return;
	temp_1[h-0] = A[i][j][h] * C4[h][k];
}

__global__ void kernel_2(int i,int j){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=(900-1) ) )return;
	int thread_count_1 = (900-1)-0+1;
	float* temp_1 = (float*)malloc(sizeof(float)*((900-1)-0+1));
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>(i,j,k,temp_1);
	hipDeviceSynchronize();
	sumArray( temp_1,thread_count_1);
	hipDeviceSynchronize();
	sum_[k] = temp_1[0];
}

__global__ void kernel_3(int i,int j){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=(900-1) ) )return;
	A[i][j][k] = sum_[k];
}

__global__ void kernel_4(int i){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(800-1) ) )return;
	int thread_count_2 = (900-1)-0+1;
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>(i,j);
	hipDeviceSynchronize();
	int thread_count_3 = (900-1)-0+1;
	kernel_3<<<ceil( 1.0 * thread_count_3/1024),1024>>>(i,j);
	hipDeviceSynchronize();
}

__global__ void kernel_5(){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=(700-1) ) )return;
	int thread_count_4 = (800-1)-0+1;
	kernel_4<<<ceil( 1.0 * thread_count_4/1024),1024>>>(i);
	hipDeviceSynchronize();
}

__global__ void main_kernel(){
	int thread_count_5 = (700-1)-0+1;
	kernel_5<<<ceil( 1.0 * thread_count_5/1024),1024>>>();
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_A = (float*) malloc(sizeof(float)* (702)* (802)* (902));
	hipMemcpyFromSymbol(h_A,HIP_SYMBOL(A),sizeof(float)* (702)* (802)* (902));
	float* h_C4 = (float*) malloc(sizeof(float)* (902)* (902));
	hipMemcpyFromSymbol(h_C4,HIP_SYMBOL(C4),sizeof(float)* (902)* (902));
	float* h_sum_ = (float*) malloc(sizeof(float)* (902));
	hipMemcpyFromSymbol(h_sum_,HIP_SYMBOL(sum_),sizeof(float)* (902));
	gettimeofday(&t2, 0);
	double time = 1.0*(t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time taken for execution is: %.8f sec\n", time);
	return 0;
}
