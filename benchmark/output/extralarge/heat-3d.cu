#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

__device__ float A[203][203][203];
__device__ float B[203][203][203];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(int i,int j){
	int k = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=k ) || !( k<=(200-1) ) )return;
	
	
	
	B[i][j][k] = A[i][j][k] + 0.125 * (A[i+1][j][k] - 2.0 * A[i][j][k] + A[i-1][j][k])+0.125 * (A[i][j+1][k] - 2.0 * A[i][j][k] + A[i][j-1][k])+0.125 * (A[i][j][k+1] - 2.0 * A[i][j][k] + A[i][j][k-1]);
}

__global__ void kernel_2(int i){
	int j = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=j ) || !( j<=(200-1) ) )return;
	int thread_count_1 = (200-1)-1+1;
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>(i,j);
	hipDeviceSynchronize();
}

__global__ void kernel_3(){
	int i = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=i ) || !( i<=(200-1) ) )return;
	int thread_count_2 = (200-1)-1+1;
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>(i);
	hipDeviceSynchronize();
}

__global__ void kernel_4(int i,int j){
	int k = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=k ) || !( k<=(200-1) ) )return;
	
	
	
	A[i][j][k] = B[i][j][k] + 0.125 * (B[i+1][j][k] - 2.0 * B[i][j][k] + B[i-1][j][k])+0.125 * (B[i][j+1][k] - 2.0 * B[i][j][k] + B[i][j-1][k])+0.125 * (B[i][j][k+1] - 2.0 * B[i][j][k] + B[i][j][k-1]);
}

__global__ void kernel_5(int i){
	int j = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=j ) || !( j<=(200-1) ) )return;
	int thread_count_4 = (200-1)-1+1;
	kernel_4<<<ceil( 1.0 * thread_count_4/1024),1024>>>(i,j);
	hipDeviceSynchronize();
}

__global__ void kernel_6(){
	int i = 1 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 1<=i ) || !( i<=(200-1) ) )return;
	int thread_count_5 = (200-1)-1+1;
	kernel_5<<<ceil( 1.0 * thread_count_5/1024),1024>>>(i);
	hipDeviceSynchronize();
}

__global__ void kernel_7(){
	int t = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=t ) || !( t<=1000 ) )return;
	int thread_count_3 = (200-1)-1+1;
	kernel_3<<<ceil( 1.0 * thread_count_3/1024),1024>>>();
	hipDeviceSynchronize();
	int thread_count_6 = (200-1)-1+1;
	kernel_6<<<ceil( 1.0 * thread_count_6/1024),1024>>>();
	hipDeviceSynchronize();
}

__global__ void main_kernel(){
	int thread_count_7 = 1000-0+1;
	kernel_7<<<ceil( 1.0 * thread_count_7/1024),1024>>>();
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_A = (float*) malloc(sizeof(float)* (203)* (203)* (203));
	hipMemcpyFromSymbol(h_A,HIP_SYMBOL(A),sizeof(float)* (203)* (203)* (203));
	float* h_B = (float*) malloc(sizeof(float)* (203)* (203)* (203));
	hipMemcpyFromSymbol(h_B,HIP_SYMBOL(B),sizeof(float)* (203)* (203)* (203));
	gettimeofday(&t2, 0);
	double time = 1.0*(t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time taken for execution is: %.8f sec\n", time);
	return 0;
}
