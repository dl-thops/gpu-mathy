#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<math.h>
#include <sys/time.h>

__device__ float A[1802][2202];
__device__ float tmp[1802];
__device__ float x[2202];
__device__ float y[1802];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(int i,float* temp_1){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(2200-1) ) )return;
	temp_1[j-0] = A[i][j] * x[j];
}

__global__ void kernel_2(int i,float* temp_2){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=(2200-1) ) )return;
	temp_2[k-0] = A[i][k] * tmp[i];
}

__global__ void kernel_3(){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=(1800-1) ) )return;
	int thread_count_1 = (2200-1)-0+1;
	float* temp_1 = (float*)malloc(sizeof(float)*((2200-1)-0+1));
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>(i,temp_1);
	hipDeviceSynchronize();
	sumArray( temp_1,thread_count_1);
	hipDeviceSynchronize();
	tmp[i] = temp_1[0];
	int thread_count_2 = (2200-1)-0+1;
	float* temp_2 = (float*)malloc(sizeof(float)*((2200-1)-0+1));
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>(i,temp_2);
	hipDeviceSynchronize();
	sumArray( temp_2,thread_count_2);
	hipDeviceSynchronize();
	y[i] = temp_2[0];
}

__global__ void main_kernel(){
	int thread_count_3 = (1800-1)-0+1;
	kernel_3<<<ceil( 1.0 * thread_count_3/1024),1024>>>();
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_A = (float*) malloc(sizeof(float)* (1802)* (2202));
	hipMemcpyFromSymbol(h_A,HIP_SYMBOL(A),sizeof(float)* (1802)* (2202));
	float* h_tmp = (float*) malloc(sizeof(float)* (1802));
	hipMemcpyFromSymbol(h_tmp,HIP_SYMBOL(tmp),sizeof(float)* (1802));
	float* h_x = (float*) malloc(sizeof(float)* (2202));
	hipMemcpyFromSymbol(h_x,HIP_SYMBOL(x),sizeof(float)* (2202));
	float* h_y = (float*) malloc(sizeof(float)* (1802));
	hipMemcpyFromSymbol(h_y,HIP_SYMBOL(y),sizeof(float)* (1802));
	gettimeofday(&t2, 0);
	double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
	printf("Time taken for execution is: %.6f ms\n", time);
	return 0;
}
