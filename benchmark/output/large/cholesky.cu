#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

__device__ float A[2002][2002];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(int i,int j){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=(j-1) ) )return;
	A[i][j] = A[i][j] - A[i][k] * A[j][k];
}

__global__ void kernel_2(int i){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(i-1) ) )return;
	int thread_count_1 = (j-1)-0+1;
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>(i,j);
	hipDeviceSynchronize();
	A[i][j] = A[i][j] / A[j][j];
}

__global__ void kernel_3(int i){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=(i-1) ) )return;
	A[i][i] = A[i][i] - A[i][k] * A[i][k];
}

__global__ void kernel_4(){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=(2000-1) ) )return;
	int thread_count_2 = (i-1)-0+1;
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>(i);
	hipDeviceSynchronize();
	int thread_count_3 = (i-1)-0+1;
	kernel_3<<<ceil( 1.0 * thread_count_3/1024),1024>>>(i);
	hipDeviceSynchronize();
	A[i][i] = sqrt(A[i][i]);
}

__global__ void main_kernel(){
	int thread_count_4 = (2000-1)-0+1;
	kernel_4<<<ceil( 1.0 * thread_count_4/1024),1024>>>();
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_A = (float*) malloc(sizeof(float)* (2002)* (2002));
	hipMemcpyFromSymbol(h_A,HIP_SYMBOL(A),sizeof(float)* (2002)* (2002));
	gettimeofday(&t2, 0);
	double time = 1.0*(t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time taken for execution is: %.8f sec\n", time);
	return 0;
}
