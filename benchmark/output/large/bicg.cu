#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

__device__ float A[2202][1802];
__device__ float p[1802];
__device__ float q[2202];
__device__ float r[2202];
__device__ float s[2202];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(int i,float* temp_1){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(1800-1) ) )return;
	temp_1[j-0] = A[i][j] * r[i];
}

__global__ void kernel_2(int i,float* temp_2){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(1800-1) ) )return;
	temp_2[j-0] = A[i][j] * p[j];
}

__global__ void kernel_3(){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=(2200-1) ) )return;
	int thread_count_1 = (1800-1)-0+1;
	float* temp_1 = (float*)malloc(sizeof(float)*((1800-1)-0+1));
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>(i,temp_1);
	hipDeviceSynchronize();
	sumArray( temp_1,thread_count_1);
	hipDeviceSynchronize();
	s[i] = temp_1[0];
	int thread_count_2 = (1800-1)-0+1;
	float* temp_2 = (float*)malloc(sizeof(float)*((1800-1)-0+1));
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>(i,temp_2);
	hipDeviceSynchronize();
	sumArray( temp_2,thread_count_2);
	hipDeviceSynchronize();
	q[i] = temp_2[0];
}

__global__ void main_kernel(){
	int thread_count_3 = (2200-1)-0+1;
	kernel_3<<<ceil( 1.0 * thread_count_3/1024),1024>>>();
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_A = (float*) malloc(sizeof(float)* (2202)* (1802));
	hipMemcpyFromSymbol(h_A,HIP_SYMBOL(A),sizeof(float)* (2202)* (1802));
	float* h_p = (float*) malloc(sizeof(float)* (1802));
	hipMemcpyFromSymbol(h_p,HIP_SYMBOL(p),sizeof(float)* (1802));
	float* h_q = (float*) malloc(sizeof(float)* (2202));
	hipMemcpyFromSymbol(h_q,HIP_SYMBOL(q),sizeof(float)* (2202));
	float* h_r = (float*) malloc(sizeof(float)* (2202));
	hipMemcpyFromSymbol(h_r,HIP_SYMBOL(r),sizeof(float)* (2202));
	float* h_s = (float*) malloc(sizeof(float)* (2202));
	hipMemcpyFromSymbol(h_s,HIP_SYMBOL(s),sizeof(float)* (2202));
	gettimeofday(&t2, 0);
	double time = 1.0*(t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time taken for execution is: %.8f sec\n", time);
	return 0;
}
