#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

__device__ float A[1002][1202];
__device__ float B[1202][1102];
__device__ float C[1002][1202];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(float beta,int i){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(1100-1) ) )return;
	C[i][j] = C[i][j] * beta;
}

__global__ void kernel_2(float alpha,int i,int k,float* temp_1){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=(1100-1) ) )return;
	temp_1[j-0] = alpha * A[i][k] * B[k][j];
}

__global__ void kernel_3(float alpha,int i){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=(1200-1) ) )return;
	int thread_count_2 = (1100-1)-0+1;
	float* temp_1 = (float*)malloc(sizeof(float)*((1100-1)-0+1));
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>(alpha,i,k,temp_1);
	hipDeviceSynchronize();
	sumArray( temp_1,thread_count_2);
	hipDeviceSynchronize();
	C[i][k] = temp_1[0];
}

__global__ void kernel_4(float alpha,float beta){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=(1000-1) ) )return;
	int thread_count_1 = (1100-1)-0+1;
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>(beta,i);
	hipDeviceSynchronize();
	int thread_count_3 = (1200-1)-0+1;
	kernel_3<<<ceil( 1.0 * thread_count_3/1024),1024>>>(alpha,i);
	hipDeviceSynchronize();
}

__global__ void main_kernel(){
	float alpha;
	float beta;
	alpha = 1.5;
	beta = 1.2;
	int thread_count_4 = (1000-1)-0+1;
	kernel_4<<<ceil( 1.0 * thread_count_4/1024),1024>>>(alpha,beta);
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_A = (float*) malloc(sizeof(float)* (1002)* (1202));
	hipMemcpyFromSymbol(h_A,HIP_SYMBOL(A),sizeof(float)* (1002)* (1202));
	float* h_B = (float*) malloc(sizeof(float)* (1202)* (1102));
	hipMemcpyFromSymbol(h_B,HIP_SYMBOL(B),sizeof(float)* (1202)* (1102));
	float* h_C = (float*) malloc(sizeof(float)* (1002)* (1202));
	hipMemcpyFromSymbol(h_C,HIP_SYMBOL(C),sizeof(float)* (1002)* (1202));
	gettimeofday(&t2, 0);
	double time = 1.0*(t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time taken for execution is: %.8f sec\n", time);
	return 0;
}
