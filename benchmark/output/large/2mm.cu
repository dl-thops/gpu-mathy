#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

__device__ float a[802][1102];
__device__ float b[1102][902];
__device__ float c[902][1202];
__device__ float d[802][1202];
__device__ float temp[802][902];

__global__ void sumCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float sum = 0;
	for (int i = gthIdx; i < n; i += gridSize){
		sum += a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = sum;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] += shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void sumArray(float* a,int n) {
	sumCommMultiBlock<<<24, 1024>>>(a, n);
	sumCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void prodCommMultiBlock(float *a, int n) {
	int thIdx = threadIdx.x;
	int gthIdx = thIdx + blockIdx.x*1024;
	const int gridSize = 1024*gridDim.x;
	float prod = 1;
	for (int i = gthIdx; i < n; i += gridSize){
		prod *= a[i];
	}
	__shared__ float shArr[1024];
	shArr[thIdx] = prod;
	__syncthreads();
	for (int size = 1024/2; size>0; size/=2) {
		if (thIdx<size){
			shArr[thIdx] *= shArr[thIdx+size];
		}
		__syncthreads();
	}
	if (thIdx == 0){
		a[blockIdx.x] = shArr[0];
	}
}

__device__ void prodArray(float* a,int n) {
	prodCommMultiBlock<<<24, 1024>>>(a, n);
	prodCommMultiBlock<<<1, 1024>>>(a, 24);
	hipDeviceSynchronize();
}

__global__ void kernel_1(float alpha,int i,int j,float* temp_1){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=1100 ) )return;
	temp_1[k-0] = alpha * a[i][k] * b[k][j];
}

__global__ void kernel_2(float alpha,int i){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=900 ) )return;
	int thread_count_1 = 1100-0+1;
	float* temp_1 = (float*)malloc(sizeof(float)*(1100-0+1));
	kernel_1<<<ceil( 1.0 * thread_count_1/1024),1024>>>(alpha,i,j,temp_1);
	hipDeviceSynchronize();
	sumArray( temp_1,thread_count_1);
	hipDeviceSynchronize();
	temp[i][j] = temp_1[0];
}

__global__ void kernel_3(float alpha){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=(800-1) ) )return;
	int thread_count_2 = 900-0+1;
	kernel_2<<<ceil( 1.0 * thread_count_2/1024),1024>>>(alpha,i);
	hipDeviceSynchronize();
}

__global__ void kernel_4(int i,int j,float* temp_2){
	int k = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=k ) || !( k<=900 ) )return;
	temp_2[k-0] = temp[i][k] * c[k][j];
}

__global__ void kernel_5(float beta,int i){
	int j = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=j ) || !( j<=1200 ) )return;
	d[i][j] = d[i][j] * beta;
	int thread_count_4 = 900-0+1;
	float* temp_2 = (float*)malloc(sizeof(float)*(900-0+1));
	kernel_4<<<ceil( 1.0 * thread_count_4/1024),1024>>>(i,j,temp_2);
	hipDeviceSynchronize();
	sumArray( temp_2,thread_count_4);
	hipDeviceSynchronize();
	d[i][j] = temp_2[0];
}

__global__ void kernel_6(float beta){
	int i = 0 + blockDim.x * blockIdx.x + threadIdx.x;
	if( !( 0<=i ) || !( i<=800 ) )return;
	int thread_count_5 = 1200-0+1;
	kernel_5<<<ceil( 1.0 * thread_count_5/1024),1024>>>(beta,i);
	hipDeviceSynchronize();
}

__global__ void main_kernel(){
	float alpha;
	float beta;
	alpha = 1.5;
	beta = 1.2;
	int thread_count_3 = (800-1)-0+1;
	kernel_3<<<ceil( 1.0 * thread_count_3/1024),1024>>>(alpha);
	hipDeviceSynchronize();
	int thread_count_6 = 800-0+1;
	kernel_6<<<ceil( 1.0 * thread_count_6/1024),1024>>>(beta);
	hipDeviceSynchronize();
	return;
}

int main(){
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	main_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float* h_a = (float*) malloc(sizeof(float)* (802)* (1102));
	hipMemcpyFromSymbol(h_a,HIP_SYMBOL(a),sizeof(float)* (802)* (1102));
	float* h_b = (float*) malloc(sizeof(float)* (1102)* (902));
	hipMemcpyFromSymbol(h_b,HIP_SYMBOL(b),sizeof(float)* (1102)* (902));
	float* h_c = (float*) malloc(sizeof(float)* (902)* (1202));
	hipMemcpyFromSymbol(h_c,HIP_SYMBOL(c),sizeof(float)* (902)* (1202));
	float* h_d = (float*) malloc(sizeof(float)* (802)* (1202));
	hipMemcpyFromSymbol(h_d,HIP_SYMBOL(d),sizeof(float)* (802)* (1202));
	float* h_temp = (float*) malloc(sizeof(float)* (802)* (902));
	hipMemcpyFromSymbol(h_temp,HIP_SYMBOL(temp),sizeof(float)* (802)* (902));
	gettimeofday(&t2, 0);
	double time = 1.0*(t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("Time taken for execution is: %.8f sec\n", time);
	return 0;
}
